#include "hip/hip_runtime.h"
#include "cuEntityIDBuffer.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#define nTPB 256
#include <stdio.h>
//#pragma once


__global__ void caculateKernel(cuEntityIDBuffer** d_cuEntityIDList, int listSize, unsigned int* res)
{
	int idx=blockDim.x * blockIdx.x + threadIdx.x;
	for(unsigned i = 0; i < listSize; ++i) {
		/* code */
		//(d_cuEntityIDList[3])->cuBuffer[idx]+=(d_cuEntityIDList[i])->cuBuffer[idx];
		//unsigned int* temptr = (d_cuEntityIDList[3])->getcuEntityIDBuffer();
		unsigned int* temptr_i = (d_cuEntityIDList[i])->getcuEntityIDBuffer();
		//temptr[idx] += temptr_i[idx];
		res[idx]+=temptr_i[idx];
	}
}

int main(int argc, char const *argv[])
{
	unsigned int* h_buf, *h_buf1, *h_buf2;
	unsigned int* h_res;

	h_buf=(unsigned int *)(malloc(1024*sizeof(unsigned int)));
	h_buf1=(unsigned int *)(malloc(1024*sizeof(unsigned int)));
	h_buf2=(unsigned int *)(malloc(1024*sizeof(unsigned int)));
	h_res=(unsigned int *)(malloc(1024*sizeof(unsigned int)));
	for(unsigned int i = 0; i < 1024; ++i) {
		/* code */
		h_buf[i]=i;
		h_buf1[i]=i;
		h_buf2[i]=i;
		h_res[i]=0;
	}
	for(unsigned int i = 0; i < 1024; ++i) {
		/* code */
		printf("%u ",h_buf[i]);
	}

	cuEntityIDBuffer d_buf(h_buf,1024);
	cuEntityIDBuffer d_buf1(h_buf1,1024);
	cuEntityIDBuffer d_buf2(h_buf2,1024);
	//cuEntityIDBuffer d_res(h_res,1024);

	unsigned int* d_res;
	hipMalloc((void **)&d_res, 1024 * sizeof(unsigned int));
	hipMemcpy(d_res,h_res,1024*sizeof(unsigned int),hipMemcpyHostToDevice);


	thrust::host_vector<cuEntityIDBuffer*> h_BufferList(4);
	h_BufferList.push_back(&d_buf);
	h_BufferList.push_back(&d_buf1);
	h_BufferList.push_back(&d_buf2);
	//h_BufferList.push_back(&d_res);
	thrust::device_vector<cuEntityIDBuffer*> cuBufferList = h_BufferList;
	
	//CuBufferList.push_back(thrust::raw_pointer_cast(d_buf));
	//cuEntityIDBuffer** d_cuEntityIDList=thrust::raw_pointer_cast<cuEntityIDBuffer**>(cuBufferList.data()); //ask in stackover
	cuEntityIDBuffer** d_cuEntityIDList=thrust::raw_pointer_cast(cuBufferList.data());

	caculateKernel<<<((1024+nTPB-1)/nTPB),nTPB>>>(d_cuEntityIDList,cuBufferList.size(),d_res);
	hipDeviceSynchronize();
	//d_res.cuTest();
	//d_res.cuCallBackEntityIDBuffer(h_res);
	//d_res.cuCallBackEntityIDBuffer(h_res);

	hipMemcpy(h_res,d_res,1024*sizeof(unsigned int),hipMemcpyDeviceToHost);
	for(unsigned int i = 0; i < 1024; ++i) {
		/* code */
		printf("%u ",h_res[i]);
	}
	return 0;
}