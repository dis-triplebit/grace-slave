#include "hip/hip_runtime.h"
#include "cuEntityIDBuffer.h"
#include <stdio.h>
#pragma hd_warning_disable
#define nTPB 256

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void mykernel(unsigned int* buffer)
{
	int idx = threadIdx.x + (blockDim.x * blockIdx.x);
	buffer[idx]++;
}

cuEntityIDBuffer::cuEntityIDBuffer()
{
	buffersize=1024;
	gpuErrchk(hipMalloc((void **)&cuBuffer, buffersize * sizeof(unsigned int)));
}

cuEntityIDBuffer::cuEntityIDBuffer(unsigned int *buffer, int size)
{
		buffersize=size;
		gpuErrchk(hipMalloc((void **)&cuBuffer, buffersize * sizeof(unsigned int)));
		gpuErrchk(hipMemcpy(cuBuffer,buffer,buffersize*sizeof(unsigned int),hipMemcpyHostToDevice));
}

cuEntityIDBuffer::cuEntityIDBuffer(cuEntityIDBuffer* cubuffer)
{
	buffersize=cubuffer->buffersize;
	gpuErrchk(hipMalloc((void **)&cuBuffer, buffersize * sizeof(unsigned int)));
	gpuErrchk(hipMemcpy(cuBuffer,cubuffer->cuBuffer,buffersize*sizeof(unsigned int),hipMemcpyDeviceToDevice));
}

void cuEntityIDBuffer::cuCallBackEntityIDBuffer(unsigned int* buffer)
{
	//gpuErrchk(hipMemcpy(buffer,cuBuffer,buffersize*sizeof(unsigned int),hipMemcpyDeviceToHost));
}

cuEntityIDBuffer::~cuEntityIDBuffer()
{
	gpuErrchk(hipFree(cuBuffer));
}

unsigned int* cuEntityIDBuffer::getcuEntityIDBuffer()
{
	return cuBuffer;
}

void cuEntityIDBuffer::cuTest()
{
	mykernel<<<((buffersize+nTPB-1)/nTPB),nTPB>>>(cuBuffer);
	gpuErrchk(hipPeekAtLastError());
}