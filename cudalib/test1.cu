#include "hip/hip_runtime.h"
#include <stdio.h>
#include "test1.h"

#define DSIZE 1024
#define DVAL 10
#define nTPB 256

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void my_kernel1(int *data){
  int idx = threadIdx.x + (blockDim.x *blockIdx.x);
  if (idx < DSIZE) data[idx] =+ DVAL;
}

int my_test_func1(){

  int *d_data, *h_data;
  h_data = (int *) malloc(DSIZE * sizeof(int));
  if (h_data == 0) {printf("malloc fail\n"); exit(1);}
  hipMalloc((void **)&d_data, DSIZE * sizeof(int));
  cudaCheckErrors("hipMalloc fail");
  for (int i = 0; i < DSIZE; i++) h_data[i] = 0;
  hipMemcpy(d_data, h_data, DSIZE * sizeof(int), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy fail");
  my_kernel1<<<((DSIZE+nTPB-1)/nTPB), nTPB>>>(d_data);
  hipDeviceSynchronize();
  cudaCheckErrors("kernel");
  hipMemcpy(h_data, d_data, DSIZE * sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors("hipMemcpy 2");
  for (int i = 0; i < DSIZE; i++)
    if (h_data[i] != DVAL) {printf("Results check failed at offset %d, data was: %d, should be %d\n", i, h_data[i], DVAL); exit(1);}
  printf("Results check 1 passed!\n");
  return 0;
}
